#include "hip/hip_runtime.h"
// HPC133 heat diffusion equation solution
// written in thrust by matthew thoms
// next steps: CAN WE PLOT THIS IN C++?? PLEASE?? LMAO??

#include <thrust/universal_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>
#include <thrust/tuple.h>

#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <cmath>
#include <string>

#include "params.h"

// kernel for initializing position values from indices
struct posn_init {

	double x1;
	double x2;
	int maxdim;

	posn_init(double& _x1, double& _x2, int& _maxdim) {
		x1 = _x1;
		x2 = _x2;
		maxdim = _maxdim;
	}

	__host__ __device__
		double operator()(int pos) {
		return x1 + ((pos - 1) * (x2 - x1)) / maxdim;
	}
};

// kernel for initializing heat densities
struct dens_init {

	double x1;
	double x2;

	dens_init(double& _x1, double& _x2) {
		x1 = _x1;
		x2 = _x2;
	}

	__host__ __device__
		double operator()(thrust::tuple<double, double> ij) {
		double xdim = thrust::get<0>(ij);
		double ydim = thrust::get<1>(ij);
		
		double a = 1 - fabs(1 - 4 * fabs((xdim - (x1 + x2) / 2) / (x2 - x1)));
		double b = 1 - fabs(1 - 4 * fabs((ydim - (x1 + x2) / 2) / (x2 - x1)));

		return a * b;
	}
};

// kernel for heat equation timestep
struct heat_evolve {

	double D;
	double x1;
	double x2;
	double dx;
	double dt;

	heat_evolve(double& _D, double& _x1, double& _x2, double& _dx, double& _dt) {
		D = _D;
		x1 = _x1;
		x2 = _x2;
		dx = _dx;
		dt = _dt;
	}

	__host__ __device__
		double operator()(thrust::tuple<double, double, double, double, double, double, double> info) {
		double x = thrust::get<0>(info);
		double y = thrust::get<1>(info);

		if (x == x1 || x == x2 || y == x1 || y == x2) {
			return 0.0;
		}

		double top = thrust::get<2>(info);
		double left = thrust::get<3>(info);
		double center = thrust::get<4>(info);
		double right = thrust::get<5>(info);
		double bottom = thrust::get<6>(info);

		double laplacian = top + left + right + bottom - 4 * center;

		return center + (D / (dx * dx) * dt * laplacian);

	}
};

// export heat density as a .csv file
// i tried to get so many plotting libraries to function and none of them did. ouch oof owie
// plot in python for better results
void export_dens(thrust::universal_vector<double>& x,
	thrust::universal_vector<double>& y,
	thrust::universal_vector<double>& d,
	int timestep,
	int entries) {

	std::string filepath = "densdata" + std::to_string(timestep) + ".csv";
	std::ofstream densdata(filepath);

	for (int idx = 0; idx < entries; idx++) {
		densdata << x[idx] << "," << y[idx] << "," << d[idx] << "\n";
	}
}

int main() {

	// compute derived parameters
	int nrows = ((x2 - x1) / dx);
	int ncols = nrows;
	int npnts_x = ncols + 2;
	int npnts_y = nrows + 2;
	int tpnts = npnts_x * npnts_y;

	double dt = 0.25 * dx * dx / D;
	int nstep = runtime / dt;
	int nper = outtime / dt;
	if (nper == 0) {
		nper = 1;
	}

	// creating "grid" of x and y indices (cheese it with 1D vectors - numpy meshgrid approach)
	thrust::universal_vector<double> x(tpnts);
	thrust::universal_vector<double> y(tpnts);

	thrust::universal_vector<int> tr_modulo(tpnts, npnts_x);
	thrust::universal_vector<int> tr_divide(tpnts, npnts_y);
	
	thrust::sequence(x.begin(), x.end());
	thrust::sequence(y.begin(), y.end());

	thrust::transform(x.begin(), x.end(), tr_modulo.begin(), x.begin(), thrust::modulus<int>());
	thrust::transform(y.begin(), y.end(), tr_divide.begin(), y.begin(), thrust::divides<int>());

	// initialize position coordinates
	thrust::transform(x.begin(), x.end(), x.begin(), posn_init(x1, x2, ncols));
	thrust::transform(y.begin(), y.end(), y.begin(), posn_init(x1, x2, nrows));

	// initializing heat density at t = 0
	thrust::universal_vector<double> dens(tpnts, 0.0);
	thrust::universal_vector<double> densnext(tpnts, 0.0);

	thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(x.begin(), y.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(x.end(), y.end())),
		dens.begin(),
		dens_init(x1, x2));

	// create permutation iterators for laplacian (maybe find a way to simplify this... ew!)
	std::vector<int> x_vec(tpnts);
	thrust::sequence(x_vec.begin(), x_vec.end());

	std::vector<int> xmrow_vec = x_vec;
	std::rotate(xmrow_vec.begin(), xmrow_vec.begin() + npnts_x, xmrow_vec.end());
	thrust::universal_vector<int> xmrow = xmrow_vec;

	std::vector<int> xm1_vec = x_vec;
	std::rotate(xm1_vec.begin(), xm1_vec.begin() + 1, xm1_vec.end());
	thrust::universal_vector<int> xm1 = xm1_vec;

	std::vector<int> xp1_vec = x_vec;
	std::rotate(xp1_vec.rbegin(), xp1_vec.rbegin() + 1, xp1_vec.rend());
	thrust::universal_vector<int> xp1 = xp1_vec;

	std::vector<int> xprow_vec = x_vec;
	std::rotate(xprow_vec.rbegin(), xprow_vec.rbegin() + npnts_x, xprow_vec.rend());
	thrust::universal_vector<int> xprow = xprow_vec;

	auto xmrow_iter_begin = thrust::make_permutation_iterator(dens.begin(), xmrow.begin());
	auto xm1_iter_begin = thrust::make_permutation_iterator(dens.begin(), xm1.begin());
	auto xp1_iter_begin = thrust::make_permutation_iterator(dens.begin(), xp1.begin());
	auto xprow_iter_begin = thrust::make_permutation_iterator(dens.begin(), xprow.begin());

	auto xmrow_iter_end = thrust::make_permutation_iterator(dens.begin(), xmrow.end());
	auto xm1_iter_end = thrust::make_permutation_iterator(dens.begin(), xm1.end());
	auto xp1_iter_end = thrust::make_permutation_iterator(dens.begin(), xp1.end());
	auto xprow_iter_end = thrust::make_permutation_iterator(dens.begin(), xprow.end());

	// do the simulation!
	double simtime = 0.0 * dt;

	export_dens(x, y, dens, 0, tpnts);

	std::cout << "recorded initial state!\n";

	for (int s = 0; s < nstep; s++) {
		
		thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(x.begin(), y.begin(), xmrow_iter_begin, xm1_iter_begin, dens.begin(), xp1_iter_begin, xprow_iter_begin)),
			thrust::make_zip_iterator(thrust::make_tuple(x.end(), y.end(), xmrow_iter_end, xm1_iter_end, dens.end(), xp1_iter_end, xprow_iter_end)),
			densnext.begin(),
			heat_evolve(D, x1, x2, dx, dt));

		thrust::swap(dens, densnext);

		if ((s + 1) % nper == 0) {

			export_dens(x, y, dens, s, tpnts);

			std::cout << "recorded at timestep " << s << "!\n";
		}

		simtime += dt;
	}

}